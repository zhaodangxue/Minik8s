#include <iostream>
#include <hip/hip_runtime.h>

using std:: cout;

typedef struct{
    double *matriz;
    int    lin;
    int    col;
} Matriz;

__global__ void addMatrix(const Matriz A, const Matriz B, Matriz C)
{
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    int idy = threadIdx.y + blockDim.y*blockIdx.y;

    if ((idx < A.col) && (idy < A.lin))
      C.matriz[C.col*idy + idx] = A.matriz[A.col*idy + idx] + B.matriz[B.col*idy + idx];
}

void somaMatriz(const Matriz A, const Matriz B, Matriz *C)
{
    Matriz dA;
    Matriz dB;
    Matriz dC;

    int BLOCK_SIZE = 16;

    dA.lin = A.lin;
    dA.col = A.col;
    dB.lin = B.lin;
    dB.col = B.col;
    dC.lin = C->lin;
    dC.col = C->col;

    hipMalloc((void**)&dA.matriz, dA.lin*dA.col*sizeof(double));
    hipMalloc((void**)&dB.matriz, dB.lin*dB.col*sizeof(double));
    hipMalloc((void**)&dC.matriz, dC.lin*dC.col*sizeof(double));

    hipMemcpy(dA.matriz, A.matriz, dA.lin*dA.col*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dB.matriz, B.matriz, dB.lin*dB.col*sizeof(double), hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((dA.col + dimBlock.x - 1)/dimBlock.x, (dA.lin + dimBlock.y -1)/dimBlock.y);

    addMatrix<<<dimGrid, dimBlock>>>(dA, dB, dC);

    hipMemcpy(C->matriz, dC.matriz, dC.lin*dC.col*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(dA.matriz);
    hipFree(dB.matriz);
    hipFree(dC.matriz);

    return;
}

int printMatriz(Matriz *mat){
    for (int y = 0; y < mat->lin; y++)
    {
        for (int x = 0; x < mat->col; x++)
            cout << mat->matriz[y*mat->col + x] << " ";
        cout << "\n";
    }
    return 0;
}

int main(void)
{

    Matriz A;
    Matriz B;
    Matriz *C = new Matriz;
    int lin = 16;
    int col = 7;

    A.lin = lin;
    A.col = col;
    B.lin = lin;
    B.col = col;
    C->lin = lin;
    C->col = col;
    C->matriz = new double[lin*col];

    A.matriz = new double[lin*col];
    B.matriz = new double[lin*col];

    for (int y = 0; y < lin; y++)
        for (int x = 0; x < col; x++)
        {
            A.matriz[y*A.col + x] = 1./(float)(10.*x + y + 10.0);
            B.matriz[y*B.col + x] = (float)(x + y + 1);
        }
    cout << "Matriz A\n";
    printMatriz(&A);
    cout << "Matriz B\n";
    printMatriz(&B);

    somaMatriz(A, B, C);

    cout << "Matriz C\n";
    printMatriz(C);

    return 0;

}